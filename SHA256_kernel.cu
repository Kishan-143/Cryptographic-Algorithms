%%cuda --name sha.cu
#include<bits/stdc++.h>
#include "openssl/sha.h"
using namespace std;

string to_hex(unsigned char s) {
    stringstream ss;
    ss << hex << (int) s;
    return ss.str();
}   

string sha256(string line) {    
    unsigned char hash[SHA256_DIGEST_LENGTH];
    SHA256_CTX sha256;
    SHA256_Init(&sha256);
    SHA256_Update(&sha256, line.c_str(), line.length());
    SHA256_Final(hash, &sha256);
    string output = ""; 
    for (int i=0;i<SHA256_DIGEST_LENGTH;i++){
        output += to_hex(hash[i]);
    }       
    return output;
}

int main() {
    // Time Start
    string result;
    auto start = chrono::steady_clock::now();
    result=sha256("sri");
    // Time End
    auto end = chrono::steady_clock::now();
    cout << "sri\t :"<< result << endl;
    cout << "Elapsed time in nanoseconds : "<< chrono::duration_cast<chrono::nanoseconds>(end - start).count()<< " ns" << endl;
    cout << "Elapsed time in microseconds : "<< chrono::duration_cast<chrono::microseconds>(end - start).count()<< " micros" << endl;
    return 0;
}
//sri	 :d5e96656c6f455d2b0d8da4930a4c744cd86b4e6d2915de179d1f92f49316a9
//Elapsed time in nanoseconds : 68980 ns
//Elapsed time in microseconds : 68 micros