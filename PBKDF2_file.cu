%%cuda --name pbkdf2_file.cu
#include <bits/stdc++.h>
#include <locale.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <dirent.h>
#include <ctype.h>
#include <openssl/evp.h>
using namespace std;

#define KEY_LEN      32
#define KEK_KEY_LEN  32
#define ITERATION   4096 

typedef struct JOB {
  char * data;
  unsigned char* salt_value;
  unsigned char* out;
}JOB;

void pbkdf2(JOB ** jobs, int n) {    
    for(int i=0;i<n;i++){      
        unsigned char s[]={'s','a','l','t'};
      int c=PKCS5_PBKDF2_HMAC_SHA1(jobs[i]->data, strlen(jobs[i]->data), s, 4, ITERATION,KEK_KEY_LEN, jobs[i]->out);
    }
}

JOB * JOB_init(char * data, long size, char *salt,long s2) {
    JOB * j;
    j = (JOB *)malloc(sizeof(JOB));
    j->data = (char *)malloc(sizeof(char)*size);
    for(int i=0;i<size;i++) j->data[i]=data[i];
    j->salt_value= (unsigned char *)malloc(sizeof(unsigned char)*(s2));
    for(int i=0;i<s2;i++){
        j->salt_value[i]=salt[i];
    }
    j->out = (unsigned char *) malloc(sizeof(unsigned char) * KEK_KEY_LEN);    
    return j;
}
int main(int argc, char **argv) {
    
    auto start2 = chrono::steady_clock::now();
    setlocale(LC_ALL, "en_US.UTF-8");  
  	
    int n=0;
    size_t len;
    char * a_file = 0, * line = 0;
    char *buff;
    ssize_t read;
    JOB ** jobs;
    
    a_file=argv[1];
	  if (a_file) {
    		FILE * f = 0;
        f = fopen(a_file, "r");
        if(f){
            for (n = 0; getline(&line, &len, f) != -1; n++){}
            jobs = (JOB **)malloc(sizeof(JOB *)*n);
            fseek(f, 0, SEEK_SET);
            n=0;
            while(read!=-1){
                read=getline(&line,&len,f);
                if(read==-1) continue;
                char * buffer;
                long s2=4;
                buffer = (char *) malloc(sizeof(char)*(read));
                for(int i=0;i<read-1;i++){
                    buffer[i]=line[i];
                }
                jobs[n++] = JOB_init(buffer,read-1,buffer,s2);
            }
        }
	  }   

    auto start = chrono::steady_clock::now();
    pbkdf2(jobs,n);
    auto end = chrono::steady_clock::now();
    auto end2 = chrono::steady_clock::now();

    cout << "Number of tasks : "<< n<< endl;
    cout << "Avrage time for one task : "<< chrono::duration_cast<chrono::microseconds>((end - start)/n).count()<< " ms" << endl;
    cout << "Elapsed time in microseconds : "<< chrono::duration_cast<chrono::microseconds>(end - start).count()<< " micros" << endl;
    cout << "Avrage total time for one task : "<< chrono::duration_cast<chrono::nanoseconds>((end2 - start2)/n).count()<< " ns" << endl;
    cout << "Total time in microseconds : "<< chrono::duration_cast<chrono::microseconds>(end2 - start2).count()<< " micros" << endl;

    
    FILE * fp2;
    FILE * fp3;
    fp2=fopen("result_pbkdf2_cpu.txt","w");
    fp3=fopen("result_pbkdf2_cpu_detail.txt","w");
    
    if(fp2){  
        for(int j=0;j<n;j++){
           for(int i=0;i<KEK_KEY_LEN;i++) { fprintf(fp2,"%02x", jobs[j]->out[i]); }
           fprintf(fp2,"\n");     
        }
    }
	  if(fp3){
        for(int i=0;i<n;i++){
            fprintf(fp3,"Data :- %s",jobs[i]->data);
            fprintf(fp3,"len :- %d",strlen(jobs[i]->data));
            fprintf(fp3,"salt :- %s",jobs[i]->salt_value);
            fprintf(fp3,"--------\n\n");
        }   
    }
	return 0;
}
