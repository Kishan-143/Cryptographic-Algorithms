%%cuda --name sha256_cpu.cu
#include <bits/stdc++.h>
#include <locale.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include "openssl/sha.h"
#include <dirent.h>
#include <ctype.h>
using namespace std;

typedef unsigned char BYTE;             
typedef uint32_t  WORD;             
typedef struct JOB {
	BYTE * data;
	unsigned long long size;
	BYTE digest[64];
	char fname[128];
}JOB;
char * hash_to_string(BYTE * buff) {
    int k,i;
    char *res=(char *)malloc(70); 
	for (i = 0;i < 32; i++)	{
        sprintf(res + k, "%.2x", buff[i]);
        k=k+2;
	}
	res[64] = 0;
	return res;
}

void sha256(JOB ** jobs, int n) {    
    for(int i=0;i<n;i++){      
      unsigned char hash[SHA256_DIGEST_LENGTH];
      SHA256_CTX sha256;
      SHA256_Init(&sha256);
      SHA256_Update(&sha256, jobs[i]->data, jobs[i]->size);
      SHA256_Final(jobs[i]->digest, &sha256);  
    }
}

JOB * JOB_init(BYTE * data, long size, char * fname) {
    JOB * j;
    j = (JOB *)malloc(sizeof(JOB));
    j->data = (BYTE *)malloc(sizeof(BYTE)*size);
    j->data = data;
    j->size = size;
    for (int i = 0; i < 64; i++){
        j->digest[i] = 0xff;
    }
    strcpy(j->fname, fname);
    return j;
}

int main(int argc, char **argv) {
    auto start2 = chrono::steady_clock::now();

    setlocale(LC_ALL, "en_US.UTF-8");  
	int i = 0, n = 0;
	size_t len;
	unsigned long temp;
	char * a_file = 0, * line = 0;
	BYTE * buff;
	char option, index;
	ssize_t read;
    JOB ** jobs;
    
    a_file=argv[1];
	if (a_file) {
		FILE * f = 0;
        f = fopen(a_file, "r");
        if(f){
            for (n = 0; getline(&line, &len, f) != -1; n++){}
            jobs = (JOB **)malloc(sizeof(JOB *)*n);
            fseek(f, 0, SEEK_SET);
            n=0;
            while(read!=-1){
                    read=getline(&line,&len,f);
                    if(read==-1) continue;
                    BYTE * buffer;
                    buffer = (BYTE *) malloc(sizeof(char)*(read));
                    for(int i=0;i<read-1;i++){
                    buffer[i]=line[i];
                }
                jobs[n++] = JOB_init(buffer,read-1,line);
            }
        }
	} 

    auto start = chrono::steady_clock::now();
    sha256(jobs,n); 
    auto end = chrono::steady_clock::now();
    auto end2 = chrono::steady_clock::now();

    cout << "Number of tasks : "<< n<< endl;
    cout << "Avrage time for one task : "<< chrono::duration_cast<chrono::nanoseconds>((end - start)/n).count()<< " ns" << endl;
    cout << "Elapsed time in nanoseconds : "<< chrono::duration_cast<chrono::nanoseconds>(end - start).count()<< " ns" << endl;
    cout << "Elapsed time in microseconds : "<< chrono::duration_cast<chrono::microseconds>(end - start).count()<< " micros" << endl;
    cout << "Avrage total time for one task : "<< chrono::duration_cast<chrono::nanoseconds>((end2 - start2)/n).count()<< " ns" << endl;
    cout << "Total time in nanoseconds : "<< chrono::duration_cast<chrono::nanoseconds>(end2 - start2).count()<< " ns" << endl;
    cout << "Total time in microseconds : "<< chrono::duration_cast<chrono::microseconds>(end2 - start2).count()<< " micros" << endl;

    FILE * fp2;
    FILE * fp3;
    fp2=fopen("result_sha256_cpu.txt","w");
    fp3=fopen("result_sha256_cpu_detail.txt","w");
    
    if(fp2){
        for(int i=0;i<n;i++){
            fprintf(fp2,"%s\n",hash_to_string(jobs[i]->digest));
        }
    }
	if(fp3){
        for(int i=0;i<n;i++){
            fprintf(fp3,"Data :- %s",jobs[i]->data);
            fprintf(fp3,"Hashing String :- %s",jobs[i]->digest);
            fprintf(fp3,"Hashing digest :- %s\n",hash_to_string(jobs[i]->digest));
            fprintf(fp3,"--------\n\n");
        }   
    }
	return 0;
}

/*
Number of tasks : 230450
Avrage time for one task : 245 ns
Elapsed time in nanoseconds : 56634567 ns
Elapsed time in microseconds : 56634 micros
Avrage total time for one task : 570 ns
Total time in nanoseconds : 131569122 ns
Total time in microseconds : 131569 micros
*/