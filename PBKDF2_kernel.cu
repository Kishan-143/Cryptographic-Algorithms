%%cuda --name pbkdf2_kernel.cu
#include<bits/stdc++.h>
#include <openssl/evp.h>
using namespace std;

#define KEY_LEN      64
#define KEK_KEY_LEN  20
#define ITERATION   4096 
int main()
{ 
    size_t i;
    unsigned char *out;
    const char pwd[] = "password";
    unsigned char salt_value[] = {'s','a','l','t'};
    out = (unsigned char *) malloc(sizeof(unsigned char) * KEK_KEY_LEN);
    
    auto start = chrono::steady_clock::now();
    int c=PKCS5_PBKDF2_HMAC_SHA1(pwd, strlen(pwd), salt_value, sizeof(salt_value), ITERATION,KEK_KEY_LEN, out);
    auto end = chrono::steady_clock::now();
    cout<<"Password : "<<pwd<<endl;
    cout<<"Iterations : "<<ITERATION<<endl;
    cout<<"Salt :"; for(i=0;i<sizeof(salt_value);i++) { printf("%02x", salt_value[i]); } cout<<endl;
    cout<<"Hash :"; for(i=0;i<KEK_KEY_LEN;i++) { printf("%02x", out[i]); } printf("\n");    
    cout << "Elapsed time in nanoseconds : "<< chrono::duration_cast<chrono::nanoseconds>(end - start).count()<< " ns" << endl;
    cout << "Elapsed time in microseconds : "<< chrono::duration_cast<chrono::microseconds>(end - start).count()<< " micros" << endl;
    return 0;
}

//Password : password
//Iterations : 4096
//Salt :73616c74
//Hash :4b007901b765489abead49d926f721d065a429c1
//Elapsed time in nanoseconds : 2006218 ns
//Elapsed time in microseconds : 2006 micros